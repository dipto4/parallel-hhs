#include<iostream>
#include<cstdlib>
#include<cassert>
#include<vector>
#include<fstream>
#include<ctime>

#include "harness.h"
#include "../scan_optimized_1.cuh"
#include "../cpu_scan.h"



/*template<int p>
  struct createIC {
  constexpr createIC() : arr() {
  arr[0] = 1;
  for(int i = 1; i <= p ; i++) {
  arr[i] = arr[i-1] * 2;
  }
  }
  int arr[p+1];
  };*/


typedef float real_t;

template<int NSIZE, int BLOCKSIZE>
void perf_split_gpu_warm() {
    real_t *total;
    real_t *slow, *fast;
    real_t *d_slow, *d_fast, *d_total;
    int* d_predicate; int* d_scanned_predicate;

    partition* sizes = new partition();
    partition* d_sizes;

    const size_t N = NSIZE;

    // generate the random numbers first

    slow = new real_t[N];
    fast = new real_t[N];
    total = new real_t[N];

    //const real_t lo = 0.;
    const real_t hi = 10.0;

    for(int i = 0 ; i < N; i++) {
        float r = static_cast<float> (rand()) / static_cast<float>(RAND_MAX); 

        total[i] = hi * r; 
    }

    const real_t pivot = 5.0;

    // allocate resources on the GPU

    hipMalloc(&d_slow, sizeof(real_t) * N);
    hipMalloc(&d_fast, sizeof(real_t) * N);
    hipMalloc(&d_total, sizeof(real_t) * N);
    hipMalloc(&d_sizes, sizeof(partition));
    hipMalloc(&d_predicate, sizeof(int) * N);
    hipMalloc(&d_scanned_predicate, sizeof(int) * N);


    hipMemcpy(d_total, total, sizeof(real_t) * N , hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    split<real_t, BLOCKSIZE>(d_total, d_slow, d_fast, d_predicate, d_scanned_predicate, pivot, N, d_sizes); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;

    hipEventElapsedTime(&time, start, stop);

    std::cout<<"GPU elapsed time: "<<time<<std::endl;

    hipMemcpy(slow, d_slow, sizeof(real_t) * N , hipMemcpyDeviceToHost);
    hipMemcpy(fast, d_fast, sizeof(real_t) * N, hipMemcpyDeviceToHost);
    hipMemcpy(sizes, d_sizes, sizeof(partition), hipMemcpyDeviceToHost);


    hipFree(d_slow);
    hipFree(d_fast);
    hipFree(d_total);

    delete [] slow;
    delete [] fast;
    delete [] total;

}

template<int NSIZE, int BLOCKSIZE, int NEVENTS>
void perf_split_gpu(std::ofstream &outfile) {
    outfile<<NSIZE<<"\t";
    //float avg_time = 0;
    for(int event = 0 ; event < NEVENTS; event++) {
        srand(time(0));
        real_t *total;
        real_t *slow, *fast;
        real_t *d_slow, *d_fast, *d_total;
        int* d_predicate; int* d_scanned_predicate;

        partition* sizes = new partition();
        partition* d_sizes;

        const size_t N = NSIZE;

        // generate the random numbers first

        slow = new real_t[N];
        fast = new real_t[N];
        total = new real_t[N];

        //const real_t lo = 0.;
        const real_t hi = 10.0;

        for(int i = 0 ; i < N; i++) {
            float r = static_cast<float> (rand()) / static_cast<float>(RAND_MAX); 

            total[i] = hi * r; 
        }

        const real_t pivot = 5.0;

        // allocate resources on the GPU

        hipMalloc(&d_slow, sizeof(real_t) * N);
        hipMalloc(&d_fast, sizeof(real_t) * N);
        hipMalloc(&d_total, sizeof(real_t) * N);
        hipMalloc(&d_sizes, sizeof(partition));
        hipMalloc(&d_predicate, sizeof(int) * N);
        hipMalloc(&d_scanned_predicate, sizeof(int) * N);


        hipMemcpy(d_total, total, sizeof(real_t) * N , hipMemcpyHostToDevice);

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        split<real_t, BLOCKSIZE>(d_total, d_slow, d_fast, d_predicate, d_scanned_predicate, pivot, N, d_sizes); 
        //hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float time = 0;

        hipEventElapsedTime(&time, start, stop);
        outfile<<time<<"\t";
        //avg_time += time;

        hipMemcpy(slow, d_slow, sizeof(real_t) * N , hipMemcpyDeviceToHost);
        hipMemcpy(fast, d_fast, sizeof(real_t) * N, hipMemcpyDeviceToHost);
        hipMemcpy(sizes, d_sizes, sizeof(partition), hipMemcpyDeviceToHost);


        hipFree(d_slow);
        hipFree(d_fast);
        hipFree(d_total);

        delete [] slow;
        delete [] fast;
        delete [] total;

    }
    outfile<<std::endl;


}





template<int NSIZE, int NEVENTS>
void perf_split_cpu(std::ofstream &outfile) {
    float avg_time = 0.0;
    outfile<<NSIZE<<"\t";
    for(int event = 0; event < NEVENTS; event++) {

        srand(time(0));
        real_t *total;
        real_t *slow, *fast;

        partition_c* sizes = new partition_c();

        const size_t N = NSIZE;

        // generate the random numbers first

        total = new real_t[N];

        //const real_t lo = 0.;
        const real_t hi = 10.0;

        for(int i = 0 ; i < N; i++) {
            float r = static_cast<float> (rand()) / static_cast<float>(RAND_MAX); 

            total[i] = hi * r; 
        }

        const real_t pivot = 5.0;

        // allocate resources on the GPU


        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        cpu_split<real_t>(total, slow, fast, pivot,sizes, N);
        //split<real_t, BLOCKSIZE>(d_total, d_slow, d_fast, d_predicate, d_scanned_predicate, pivot, N, d_sizes); 



        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time = 0;

        hipEventElapsedTime(&time, start, stop);
        outfile<<time<<"\t";
        //avg_time += time;
        delete [] total;


    }
    outfile<<std::endl;


}





int main() {

    std::cout<<"GPU warmup routine! IGNORE THIS CASE"<<std::endl;
    perf_split_gpu_warm<524288,128>();
    /* 
       constexpr auto IC_N = createIC<22>();

       std::ofstream outCPUFile("cpu_split.txt");
       std::ofstream outGPUFile("gpu_split_0.txt");

       for(const auto &N : IC_N.arr) {
       perf_split_gpu<N,128>(outGPUFile);
       perf_split_cpu<N>(outCPUFile);
       }

     */
    std::ofstream outCPUFile("cpu_split.txt");
    std::ofstream outGPUFile("gpu_split_2.txt");


    perf_split_gpu<1024,1024, 10>(outGPUFile);
    perf_split_cpu<1024,10>(outCPUFile);

    perf_split_gpu<2048,1024, 10>(outGPUFile);
    perf_split_cpu<2048,10>(outCPUFile);

    perf_split_gpu<4096,1024, 10>(outGPUFile);
    perf_split_cpu<4096,10>(outCPUFile);

    perf_split_gpu<8192,512, 10>(outGPUFile);
    perf_split_cpu<8192,10>(outCPUFile);

    perf_split_gpu<16384,512, 10>(outGPUFile);
    perf_split_cpu<16384,10>(outCPUFile);

    perf_split_gpu<32768,512, 10>(outGPUFile);
    perf_split_cpu<32768,10>(outCPUFile);

    perf_split_gpu<65536,256, 10>(outGPUFile);
    perf_split_cpu<65536,10>(outCPUFile);

    perf_split_gpu<131072,256, 10>(outGPUFile);
    perf_split_cpu<131072,10>(outCPUFile);

    perf_split_gpu<262144,256, 10>(outGPUFile);
    perf_split_cpu<262144,10>(outCPUFile);

    perf_split_gpu<524288,128, 10>(outGPUFile);
    perf_split_cpu<524288,10>(outCPUFile);

    perf_split_gpu<524288*2,128, 10>(outGPUFile);
    perf_split_cpu<524288*2,10>(outCPUFile);

    perf_split_gpu<524288*2*2,128, 10>(outGPUFile);
    perf_split_cpu<524288*2*2,10>(outCPUFile);

    perf_split_gpu<524288*2*2*2,128, 10>(outGPUFile);
    perf_split_cpu<524288*2*2*2,10>(outCPUFile);





    outCPUFile.close();
    outGPUFile.close();

}
